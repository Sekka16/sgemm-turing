// smem_limits_min.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define CHECK(x)                                                                                   \
    do                                                                                             \
    {                                                                                              \
        auto e = (x);                                                                              \
        if (e != hipSuccess)                                                                      \
        {                                                                                          \
            fprintf(stderr, "CUDA error %s @ %s:%d\n", hipGetErrorString(e), __FILE__, __LINE__); \
            exit(1);                                                                               \
        }                                                                                          \
    } while (0)

__global__ void dummy_kernel() {}

int main()
{
    int dev = 0;
    CHECK(hipGetDevice(&dev));

    // 必要的上限（都应该支持）
    int per_sm = 0, per_block_default = 0, per_block_optin = 0;
    CHECK(hipDeviceGetAttribute(&per_sm, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev));
    CHECK(hipDeviceGetAttribute(&per_block_default, hipDeviceAttributeMaxSharedMemoryPerBlock, dev));
    // 旧驱动可能不支持 optin 查询：做兼容处理
    hipError_t e = hipDeviceGetAttribute(&per_block_optin, hipDeviceAttributeSharedMemPerBlockOptin, dev);
    if (e != hipSuccess)
    {
        per_block_optin = per_block_default; // 退化为默认上限
    }

    hipDeviceProp_t prop{};
    CHECK(hipGetDeviceProperties(&prop, dev));
    printf("Device: %s (CC %d.%d)\n", prop.name, prop.major, prop.minor);
    printf("Shared mem per SM (hard cap): %d bytes\n", per_sm);
    printf("Per-block shared mem default : %d bytes\n", per_block_default);
    printf("Per-block shared mem opt-in  : %d bytes\n", per_block_optin);

    // 读取 kernel 的动态共享内存默认上限
    hipFuncAttributes attr{};
    CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(dummy_kernel)));
    printf("\n[kernel] default max dynamic smem: %d bytes\n", attr.maxDynamicSharedSizeBytes);

    // 尝试把该 kernel 的动态共享内存上限提升到 opt-in 值（若不支持会返回错误）
    if (per_block_optin > per_block_default)
    {
        e = hipFuncSetAttribute(reinterpret_cast<const void*>(dummy_kernel),
                                 hipFuncAttributeMaxDynamicSharedMemorySize, per_block_optin);
        printf("opt-in set to %d bytes: %s\n", per_block_optin,
               (e == hipSuccess ? "OK" : hipGetErrorString(e)));
        // 再读一遍看看是否生效
        CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(dummy_kernel)));
        printf("[kernel] after opt-in, max dynamic smem: %d bytes\n", attr.maxDynamicSharedSizeBytes);
    }

    // 小验证：用“当前 kernel 动态上限”发一次
    size_t dyn = (size_t)attr.maxDynamicSharedSizeBytes;
    printf("\nLaunching with dynamic smem = %zu ...\n", dyn);
    dummy_kernel<<<1, 32, dyn>>>();
    e = hipDeviceSynchronize();
    printf("%s\n", (e == hipSuccess ? "Launch OK" : hipGetErrorString(e)));
    return 0;
}
